#include <iostream>
#include <hip/hip_runtime.h>
#include <ctime>

#define BLOCK_SIZE 16  // Define BLOCK_SIZE for flexible grid size

__global__ void gameOfLife(int* A, int* B) {
    int n = 1024;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int localRow = threadIdx.y + 1;  // Adjust for shared memory padding
    int localCol = threadIdx.x + 1;

    __shared__ int sharedA[BLOCK_SIZE + 2][BLOCK_SIZE + 2];  // Shared memory with padding for halo cells

    // Initialize shared memory with padding for halos
    if (row < n && col < n) {
        // Center cell
        sharedA[localRow][localCol] = A[row * n + col];
        
        
        if (threadIdx.y == 0 && row > 0) 
            sharedA[0][localCol] = A[(row - 1) * n + col];  // Top neighbor
        if (threadIdx.y == BLOCK_SIZE - 1 && row < n - 1) 
            sharedA[BLOCK_SIZE + 1][localCol] = A[(row + 1) * n + col];  // Bottom neighbor
        if (threadIdx.x == 0 && col > 0) 
            sharedA[localRow][0] = A[row * n + (col - 1)];  // Left neighbor
        if (threadIdx.x == BLOCK_SIZE - 1 && col < n - 1) 
            sharedA[localRow][BLOCK_SIZE + 1] = A[row * n + (col + 1)];  // Right neighbor

        // Load corner halo cells only if within bounds
        if (threadIdx.x == 0 && threadIdx.y == 0 && row > 0 && col > 0) 
            sharedA[0][0] = A[(row - 1) * n + (col - 1)];  // Top-left corner
        if (threadIdx.x == BLOCK_SIZE - 1 && threadIdx.y == 0 && row > 0 && col < n - 1) 
            sharedA[0][BLOCK_SIZE + 1] = A[(row - 1) * n + (col + 1)];  // Top-right corner
        if (threadIdx.x == 0 && threadIdx.y == BLOCK_SIZE - 1 && row < n - 1 && col > 0) 
            sharedA[BLOCK_SIZE + 1][0] = A[(row + 1) * n + (col - 1)];  // Bottom-left corner
        if (threadIdx.x == BLOCK_SIZE - 1 && threadIdx.y == BLOCK_SIZE - 1 && row < n - 1 && col < n - 1) 
            sharedA[BLOCK_SIZE + 1][BLOCK_SIZE + 1] = A[(row + 1) * n + (col + 1)];  // Bottom-right corner
    }
    __syncthreads();

    
    if (row < n && col < n) {
        int liveNeighbors = 0;

        // Sum live neighbors, accounting for edges
        if (localRow > 0 && localCol > 0) liveNeighbors += sharedA[localRow - 1][localCol - 1];  // Top-left
        if (localRow > 0) liveNeighbors += sharedA[localRow - 1][localCol];  // Top
        if (localRow > 0 && localCol < BLOCK_SIZE + 1) liveNeighbors += sharedA[localRow - 1][localCol + 1];  // Top-right
        if (localCol > 0) liveNeighbors += sharedA[localRow][localCol - 1];  // Left
        if (localCol < BLOCK_SIZE + 1) liveNeighbors += sharedA[localRow][localCol + 1];  // Right
        if (localRow < BLOCK_SIZE + 1 && localCol > 0) liveNeighbors += sharedA[localRow + 1][localCol - 1];  // Bottom-left
        if (localRow < BLOCK_SIZE + 1) liveNeighbors += sharedA[localRow + 1][localCol];  // Bottom
        if (localRow < BLOCK_SIZE + 1 && localCol < BLOCK_SIZE + 1) liveNeighbors += sharedA[localRow + 1][localCol + 1];  // Bottom-right

        int currentState = sharedA[localRow][localCol];
        int nextState;

        // Game of Life rules
        if (currentState == 1) {
            nextState = (liveNeighbors == 2 || liveNeighbors == 3) ? 1 : 0;
        } else {
            nextState = (liveNeighbors == 3) ? 1 : 0;
        }

        B[row * n + col] = nextState;
    }
}

void printBoard(int* board, int startRow = 0, int startCol = 0, int rows = 10, int cols = 10) {
    int n = 1024;
    for (int i = startRow; i < startRow + rows; ++i) {
        for (int j = startCol; j < startCol + cols; ++j) {
            std::cout << board[i * n + j] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}



int main() {
    unsigned long n = 1024;
    dim3 dimGrid(n/BLOCK_SIZE,n/BLOCK_SIZE);
    dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
    

    int *A = (int*) malloc(sizeof(int)*n*n);
    int *B = (int*) malloc(sizeof(int)*n*n);
    
    // Initialize array
    int i,j;
    for (i=0; i<n; i++){
        for(j=0; j< n; j++){
            A[i*n + j]=i%2;
            B[i*n + j]=0;  
        }   
    }   
    
    int *gpu_A;
    int *gpu_B;
    
    hipMalloc((void**)&gpu_A, sizeof(int)*n*n);
    hipMalloc((void**)&gpu_B, sizeof(int)*n*n);
    struct timespec start, stop; 
    double time;
        
    int nGenerations = 1;
    printBoard(A);
    
    if( clock_gettime( CLOCK_REALTIME, &start) == -1 ) { perror( "clock gettime" );}
    hipMemcpy(gpu_A, A, sizeof(int)*n*n, hipMemcpyHostToDevice);
    for (i = 0; i < nGenerations; i++) {      
        gameOfLife<<<dimGrid,dimBlock>>>(gpu_A,gpu_B);
        hipDeviceSynchronize();
        hipMemcpy(gpu_A, gpu_B, sizeof(int) * n * n, hipMemcpyDeviceToDevice);
    }   
    hipMemcpy(B, gpu_B, sizeof(int)*n*n, hipMemcpyDeviceToHost);
    if( clock_gettime( CLOCK_REALTIME, &stop) == -1 ) { perror("clock gettime");}
    time = (stop.tv_sec - start.tv_sec)+ (double)(stop.tv_nsec - start.tv_nsec)/1e9;
    printf("time is %f ns\n", time*1e9);
    printBoard(B);
        
    hipFree(gpu_A);
    hipFree(gpu_B);
    free(A);
    free(B);
}   